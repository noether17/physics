#include "hip/hip_runtime.h"
#include "cuda_Vector.h"

// //////////////////////
// Inplace math functions
// //////////////////////

/* Replaces contents of u with *u + v.
 */
__device__
void
cu_v_add_inplace (Vector *u, Vector v)
{
    u->x += v.x;
    u->y += v.y;
    u->z += v.z;
}

/* Replaces contents of u with *u - v.
 */
__device__
void
cu_v_sub_inplace (Vector *u, Vector v)
{
    u->x -= v.x;
    u->y -= v.y;
    u->z -= v.z;
}

/* Replaces contents of u with -*u.
 */
__device__
void
cu_v_neg_inplace (Vector *u)
{
    u->x = -u->x;
    u->y = -u->y;
    u->z = -u->z;
}

/* Replaces contents of u with s*(*u).
 */
__device__
void
cu_v_scl_inplace (Coord s, Vector *u)
{
    u->x *= s;
    u->y *= s;
    u->z *= s;
}

/* Replaces contents of u with Vector in the same direction normalized
 *     to given length.
 */
__device__
void
cu_v_norm_inplace (Coord length, Vector *v)
{
    Coord _mag = cu_v_mag (*v);
    if (_mag) // do not rescale if magnitude is zero.
        cu_v_scl_inplace (length / _mag, v);
}

// ////////////////////////////////
// Math functions returning Vectors
// ////////////////////////////////

/* Returns u + v.
 */
__device__
Vector
cu_v_add (Vector u, Vector v)
{
    cu_v_add_inplace (&u, v);
    return u;
}

/* Returns u - v.
 */
__device__
Vector
cu_v_sub (Vector u, Vector v)
{
    cu_v_sub_inplace (&u, v);
    return u;
}

/* Returns -u.
 */
__device__
Vector
cu_v_neg (Vector u)
{
    cu_v_neg_inplace (&u);
    return u;
}

/* Returns s*u.
 */
__device__
Vector
cu_v_scl (Coord s, Vector u)
{
    cu_v_scl_inplace (s, &u);
    return u;
}

/* Returns Vector in the same direction as u normalized to given
 *     length.
 */
__device__
Vector
cu_v_norm (Coord length, Vector v)
{
    cu_v_norm_inplace (length, &v);
    return v;
}

/* Returns vector product of u and v.
 */
__device__
Vector
cu_v_cross (Vector u, Vector v)
{
    Coord w_x = u.y*v.z - u.z*v.y;
    Coord w_y = u.z*v.x - u.x*v.z;
    Coord w_z = u.x*v.y - u.y*v.x;
    Vector w = { w_x, w_y, w_z };
    return w;
}

// ////////////////////////////////
// Math functions returning scalars
// ////////////////////////////////

/* Returns scalar product of u and v.
 */
__device__
Coord
cu_v_dot (Vector u, Vector v)
{
    Coord product = 0.;
    product += u.x*v.x;
    product += u.y*v.y;
    product += u.z*v.z;
    return product;
}

/* Returns magnitude of u.
 */
__device__
Coord
cu_v_mag (Vector u)
{
    return sqrt (cu_v_dot (u, u));
}
